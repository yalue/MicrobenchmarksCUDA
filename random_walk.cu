// This file defines a CUDA random walk microbenchmark, which traverses an
// array in random order. This will print the times for each *block* in each
// kernel invocation. Specify the -zc command-line argument to use zero-copy
// memory.
//
// Usage: ./random_walk [-zc]

#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/mman.h>

// The seed to use for shuffling the array.
#define RANDOM_SEED (1337)

// The length, in number of 32-bit integers, of the array to walk
#define ARRAY_LENGTH (1024 * 1024 * 64)

// The number of steps in the random walk will equal these two values
// multiplied together.
#define STEPS_MAJOR (1)
#define STEPS_MINOR (ARRAY_LENGTH / 64)

// The total number of kernel invocations (measurement iterations) to perform.
#define KERNEL_INVOCATIONS (10)

// These microbenchmarks will use 2 blocks of 32 threads.
#define THREAD_COUNT (32)
#define BLOCK_COUNT (2)

// A macro which exits the program with an error message if the given value is
// not equal to cudaSuccess.
#define CheckError(val) CheckCUDAError( (val), #val, __FILE__, __LINE__ )

static void CheckCUDAError(hipError_t value, const char *function,
  const char *filename, int line) {
  if (value == hipSuccess) return;
  printf("Cuda error %d. File %s, line %d: %s\n", (int) value, filename, line,
    function);
  exit(1);
}

// Holds variables and pointers that are passed between the phases of the
// experiment. The times arrays hold the start and end time stamps for each
// block, in the order [block1_start, block1_end, block2_start, ...].
typedef struct {
  uint64_t *device_times;
  uint64_t *host_times;
  uint32_t *host_array;
  uint32_t *device_array;
  uint8_t *host_outputs;
  uint8_t *device_outputs;
  hipStream_t stream;
  // This will be nonzero if we're using zero-copy memory.
  uint8_t zero_copy;
} WalkState;

// Converts a 64-bit count of nanoseconds to a floating-point number of
// seconds.
static double ConvertToSeconds(uint64_t nanoseconds) {
  return ((double) nanoseconds) / 1e9;
}

// Returns the value of CUDA's global nanosecond timer.
static __device__ __inline__ uint64_t GlobalTimer64(void) {
  uint64_t to_return;
  asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(to_return));
  return to_return;
}

__global__ void DoRandomWalk(uint32_t *array, uint8_t *outputs,
  uint64_t *times) {
  int i, j, current, thread_number;
  uint32_t sum;
  if (threadIdx.x == 0) {
    times[blockIdx.x * 2] = GlobalTimer64();
  }
  __syncthreads();
  thread_number = (blockIdx.x * THREAD_COUNT) + threadIdx.x;
  current = thread_number;
  for (i = 0; i < STEPS_MAJOR; i++) {
    for (j = 0; j < STEPS_MINOR; j++) {
      sum += current;
      current = array[current];
    }
  }
  outputs[thread_number] = (uint8_t) sum;
  __syncthreads();
  if (threadIdx.x == 0) {
    times[(blockIdx.x * 2) + 1] = GlobalTimer64();
  }
}

// Returns a random 31-bit integer in the range [0, limit).
static inline uint32_t Rand32(uint32_t limit) {
  return lrand48() % limit;
}

// Takes an array and randomly shuffles its contents. The length parameter is
// the number of elements in the array. Won't work properly for arrays
// containing over 2^32 elements.
static void ShuffleArray(uint32_t *array, size_t length) {
  size_t i, j;
  uint32_t tmp;
  if (length <= 1) return;
  for (i = 0; i < length; i++) {
    j = i + Rand32(length - i);
    tmp = array[j];
    array[j] = array[i];
    array[i] = tmp;
  }
}

// Selects and initializes the device to run the benchmarks on.
void Initialize(int sync_level) {
  hipError_t error = hipErrorInvalidValue;
  switch (sync_level) {
  case 0:
    error = hipSetDeviceFlags(hipDeviceScheduleSpin);
    break;
  case 1:
    error = hipSetDeviceFlags(hipDeviceScheduleYield);
    break;
  case 2:
    error = hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
    break;
  default:
    printf("Invalid sync level: %d\n", sync_level);
    exit(1);
  }
  CheckError(error);
  CheckError(hipSetDevice(0));
}

// Allocates and initializes all values (including the stream), in the state
// struct. Requires the name of a file containing the pre-generated random walk
// array.
void AllocateMemory(WalkState *state) {
  uint32_t i = 0;
  size_t array_size = ARRAY_LENGTH * sizeof(uint32_t);
  size_t output_array_size = THREAD_COUNT * BLOCK_COUNT * sizeof(uint8_t);
  size_t times_array_size = BLOCK_COUNT * 2 * sizeof(uint64_t);
  CheckError(hipHostMalloc(&state->host_times, times_array_size, hipHostMallocDefault));
  CheckError(hipHostMalloc(&state->host_array, array_size, hipHostMallocDefault));
  CheckError(hipHostMalloc(&state->host_outputs, output_array_size, hipHostMallocDefault));
  if (!state->zero_copy) {
    // Should we use zero-copy for the times array? For now, we will.
    CheckError(hipMalloc(&state->device_times, times_array_size));
    CheckError(hipMalloc(&state->device_array, array_size));
    CheckError(hipMalloc(&state->device_outputs, output_array_size));
  } else {
    CheckError(hipHostGetDevicePointer(&state->device_times,
      state->host_times, 0));
    CheckError(hipHostGetDevicePointer(&state->device_array,
      state->host_array, 0));
    CheckError(hipHostGetDevicePointer(&state->device_outputs,
      state->host_outputs, 0));
  }
  CheckError(hipStreamCreate(&state->stream));
  printf("Generating random walk array... ");
  fflush(stdout);
  for (i = 0; i < ARRAY_LENGTH; i++) {
    state->host_array[i] = i;
  }
  ShuffleArray(state->host_array, ARRAY_LENGTH);
  printf("done!\n");
}

// Copies input arrays to the device. For this particular benchmark, this only
// needs to be called once, since the input array is only read, and the output
// array is always completely overwritten.
void CopyIn(WalkState *state) {
  if (state->zero_copy) return;
  size_t array_size = ARRAY_LENGTH * sizeof(uint32_t);
  CheckError(hipMemcpyAsync(state->device_array, state->host_array,
    array_size, hipMemcpyHostToDevice, state->stream));
  CheckError(hipStreamSynchronize(state->stream));
}

// Copies the output array from the device. Should be called after every
// iteration, so that times can be recorded.
void CopyOut(WalkState *state) {
  if (state->zero_copy) return;
  size_t output_array_size = THREAD_COUNT * BLOCK_COUNT * sizeof(uint8_t);
  size_t times_array_size = BLOCK_COUNT * 2 * sizeof(uint64_t);
  CheckError(hipMemcpyAsync(state->host_outputs, state->device_outputs,
    output_array_size, hipMemcpyDeviceToHost, state->stream));
  CheckError(hipMemcpyAsync(state->host_times, state->device_times,
    times_array_size, hipMemcpyDeviceToHost, state->stream));
  CheckError(hipStreamSynchronize(state->stream));
}

// Frees memory and closes the device stream. This will also reset the
// zero_copy field to 0.
void FreeMemory(WalkState *state) {
  CheckError(hipStreamSynchronize(state->stream));
  CheckError(hipStreamDestroy(state->stream));
  if (!state->zero_copy) {
    CheckError(hipFree(state->device_array));
    CheckError(hipFree(state->device_outputs));
    CheckError(hipFree(state->device_times));
  }
  CheckError(hipHostFree(state->host_array));
  CheckError(hipHostFree(state->host_outputs));
  CheckError(hipHostFree(state->host_times));
  memset(state, 0, sizeof(*state));
}

// Checks command-line arguments and sets members of the state struct if any
// are affected. May exit the program if any arguments are invalid.
static void ParseArgs(int argc, char **argv, WalkState *state) {
  int i;
  state->zero_copy = 0;
  if (argc == 1) return;
  if (argc != 2) {
    printf("Usage: %s [-zc]\n"
      "  Specify -zc to use zero-copy memory.\n", argv[0]);
    exit(1);
  }
  for (i = 1; i < argc; i++) {
    if (strncmp(argv[i], "-zc", 3) == 0) {
      state->zero_copy = 1;
      continue;
    }
    printf("Unknown argument: %s\n", argv[i]);
    exit(1);
  }
}

int main(int argc, char **argv) {
  int i, j;
  double block_start, block_end;
  WalkState state;
  srand48(RANDOM_SEED);
  ParseArgs(argc, argv, &state);
  // Initialize and allocate memory, then lock pages.
  Initialize(2);
  AllocateMemory(&state);
  if (!mlockall(MCL_CURRENT | MCL_FUTURE)) {
    printf("Error: failed locking pages in memory\n");
    return 1;
  }
  dim3 threads_per_block(THREAD_COUNT);
  dim3 block_count(BLOCK_COUNT);
  // We only need to copy in one time; the input array doesn't change.
  CopyIn(&state);
  for (i = 0; i < KERNEL_INVOCATIONS; i++) {
    DoRandomWalk<<<block_count, threads_per_block, 0, state.stream>>>(
      state.device_array, state.device_outputs, state.device_times);
    CheckError(hipStreamSynchronize(state.stream));
    CopyOut(&state);
    for (j = 0; j < BLOCK_COUNT; j++) {
      block_start = ConvertToSeconds(state.host_times[j * 2]);
      block_end = ConvertToSeconds(state.host_times[(j * 2) + 1]);
      printf("Block %d: start %f, end %f\n", j, block_start, block_end);
    }
  }
  printf("Thread outputs: ");
  for (i = 0; i < THREAD_COUNT * BLOCK_COUNT; i++) {
    printf("%d: %d, ", i, (int) (state.host_outputs[i]));
  }
  printf("\n");
  FreeMemory(&state);
  CheckError(hipDeviceReset());
  return 0;
}
